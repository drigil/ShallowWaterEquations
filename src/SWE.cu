#include "hip/hip_runtime.h"
#include "SWE.cuh"

// To Do - 
// 1. Use cuda's math.h for speedup
// 2. Check if shared memory overlaps not causing issues

// KERNELS 
// WENO reconstruction - Refer Appendix A of paper


// WENO for flux for h component and the source term derivatives  (due to the bathymetry)
__device__ void WENOPosX(float3 v_iP, float3 v_i, float3 v_iN, float b_iP, float b_i, float b_iN, float3 &flux, float &source){
	// Get the smoothness indicator
	float3 beta0 = (v_iN - v_i) * (v_iN - v_i);
	float3 beta1 = (v_i - v_iP) * (v_i - v_iP);

	v_iP.y = v_iP.y - (0.25 * GRAVITY * b_iP * b_iP);
	v_i.y = v_i.y - (0.25 * GRAVITY * b_i * b_i);
	v_iN.y = v_iN.y - (0.25 * GRAVITY * b_iN * b_iN);

	// Get the stencil approximations
	float3 flux0 = 0.5 * v_i + 0.5 * v_iN;
	float3 flux1 = -0.5 * v_iP + 1.5 * v_i;

	float source_0 = 0.5 * b_i + 0.5 * b_iN;
	float source_1 = -0.5 * b_iP + 1.5 * b_i;

	// find the non-linear weights
	float3 a0 = (2.0f / 3.0f) / ((WENO_EPSILON + beta0) * (WENO_EPSILON + beta0));
	float3 a1 = (1.0f / 3.0f) / ((WENO_EPSILON + beta1) * (WENO_EPSILON + beta1));

	float3 aTot = a0 + a1;

	float3 weight0 = a0 / aTot;
	float3 weight1 = a1 / aTot;

	flux = weight0 * flux0 + weight1 * flux1;
	source = weight0.y * source_0 + weight1.y * source_1;
}

__device__ void WENONegX(float3 v_i, float3 v_iN, float3 v_iNN, float b_i, float b_iN, float b_iNN, float3 &flux, float &source){
	// Get the smoothness indicator
	float3 beta0 = (v_iN - v_i) * (v_iN - v_i);
	float3 beta1 = (v_iNN - v_iN) * (v_iNN - v_iN);

	v_i.y = v_i.y - (0.25 * GRAVITY * b_i * b_i);
	v_iN.y = v_iN.y - (0.25 * GRAVITY * b_iN * b_iN);
	v_iNN.y = v_iNN.y - (0.25 * GRAVITY * b_iNN * b_iNN);

	// Get the stencil approximations
	float3 flux0 = 0.5 * v_iN + 0.5 * v_i;
	float3 flux1 = -0.5 * v_iNN + 1.5 * v_iN;

	float source_0 = 0.5 * b_iN + 0.5 * b_i;
	float source_1 = -0.5 * b_iNN + 1.5 * b_iN;

	// find the non-linear weights
	float3 a0 = (2.0f / 3.0f) / ((WENO_EPSILON + beta0) * (WENO_EPSILON + beta0));
	float3 a1 = (1.0f / 3.0f) / ((WENO_EPSILON + beta1) * (WENO_EPSILON + beta1));

	float3 aTot = a0 + a1;

	float3 weight0 = a0 / aTot;
	float3 weight1 = a1 / aTot;

	flux = weight0 * flux0 + weight1 * flux1;
	source = weight0.y * source_0 + weight1.y * source_1;
}

__device__ void WENOPosY(float3 v_iP, float3 v_i, float3 v_iN, float b_iP, float b_i, float b_iN, float3 &flux, float &source){
	// Get the smoothness indicator
	float3 beta0 = (v_iN - v_i) * (v_iN - v_i);
	float3 beta1 = (v_i - v_iP) * (v_i - v_iP);

	v_iP.z = v_iP.z - (0.25 * GRAVITY * b_iP * b_iP);
	v_i.z = v_i.z - (0.25 * GRAVITY * b_i * b_i);
	v_iN.z = v_iN.z - (0.25 * GRAVITY * b_iN * b_iN);

	// Get the stencil approximations
	float3 flux0 = 0.5 * v_i + 0.5 * v_iN;
	float3 flux1 = -0.5 * v_iP + 1.5 * v_i;

	float source_0 = 0.5 * b_i + 0.5 * b_iN;
	float source_1 = -0.5 * b_iP + 1.5 * b_i;

	// find the non-linear weights
	float3 a0 = (2.0f / 3.0f) / ((WENO_EPSILON + beta0) * (WENO_EPSILON + beta0));
	float3 a1 = (1.0f / 3.0f) / ((WENO_EPSILON + beta1) * (WENO_EPSILON + beta1));

	float3 aTot = a0 + a1;

	float3 weight0 = a0 / aTot;
	float3 weight1 = a1 / aTot;

	flux = weight0 * flux0 + weight1 * flux1;
	source = weight0.z * source_0 + weight1.z * source_1;
}

__device__ void WENONegY(float3 v_i, float3 v_iN, float3 v_iNN, float b_i, float b_iN, float b_iNN, float3 &flux, float &source){
	// Get the smoothness indicator
	float3 beta0 = (v_iN - v_i) * (v_iN - v_i);
	float3 beta1 = (v_iNN - v_iN) * (v_iNN - v_iN);

	v_i.z = v_i.z - (0.25 * GRAVITY * b_i * b_i);
	v_iN.z = v_iN.z - (0.25 * GRAVITY * b_iN * b_iN);
	v_iNN.z = v_iNN.z - (0.25 * GRAVITY * b_iNN * b_iNN);

	// Get the stencil approximations
	float3 flux0 = 0.5 * v_iN + 0.5 * v_i;
	float3 flux1 = -0.5 * v_iNN + 1.5 * v_iN;

	float source_0 = 0.5 * b_iN + 0.5 * b_i;
	float source_1 = -0.5 * b_iNN + 1.5 * b_iN;

	// find the non-linear weights
	float3 a0 = (2.0f / 3.0f) / ((WENO_EPSILON + beta0) * (WENO_EPSILON + beta0));
	float3 a1 = (1.0f / 3.0f) / ((WENO_EPSILON + beta1) * (WENO_EPSILON + beta1));

	float3 aTot = a0 + a1;

	float3 weight0 = a0 / aTot;
	float3 weight1 = a1 / aTot;

	flux = weight0 * flux0 + weight1 * flux1;
	source = weight0.z * source_0 + weight1.z * source_1;
}


// Solve Shallow Water Equations
__global__ void applySWE(float* d_height, float* d_momentumU, float* d_momentumV, int* d_offsetX, int* d_offsetY){

	// Should total be 40.96KB for 32 X 32
	__shared__ float terrainArr[NUM_THREADS_Y][NUM_THREADS_X]; //Contains terrain point heights
	__shared__ float3 pointInfoArr[NUM_THREADS_Y][NUM_THREADS_X]; // Contains height and Eigen values 
	__shared__ float3 fluxFArr[NUM_THREADS_Y][NUM_THREADS_X]; // Contains F Flux values
	__shared__ float3 fluxGArr[NUM_THREADS_Y][NUM_THREADS_X]; // Contains G flux values

	// Getting global threadIDs
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;

	float height = d_height[x + y*gridDim.x*blockDim.x];
	float momentumU = (height != 0.0) ? d_momentumU[x + y*gridDim.x*blockDim.x] / height : 0.0; 
	float momentumV = (height != 0.0) ? d_momentumV[x + y*gridDim.x*blockDim.x] / height : 0.0;

	int offsetX = d_offsetX[x + y*gridDim.x*blockDim.x];
	int offsetY = d_offsetY[x + y*gridDim.x*blockDim.x];

	// Used to compute alpha
	float eigenX = abs(momentumU) + sqrt(GRAVITY * height);
	float eigenY = abs(momentumV) + sqrt(GRAVITY * height);

	float bathymetryVal = 1.0f; // Implement later using height map

	terrainArr[threadIdx.y][threadIdx.x] = bathymetryVal;
	pointInfoArr[threadIdx.y][threadIdx.x] = make_float3(height, eigenX, eigenY);

	__syncthreads(); // Fill block of threads with terrain heights, current point height and eigen values

	// Get corresponding local alphas from obtained eigen values
	float alphaX = eigenX;
	float alphaY = eigenY;

	for(int k = -1; k<=2; k++){
		alphaX = max(alphaX, pointInfoArr[threadIdx.y][threadIdx.x + k].y);
		alphaY = max(alphaY, pointInfoArr[threadIdx.y + k][threadIdx.x].z);	
	}

	// Find the 2nd part of the low-order Lax Friedrich flux for the h component
	float hFluxX = -0.5 * alphaX * (pointInfoArr[threadIdx.y][threadIdx.x + 1].x - height);
	float hFluxY = -0.5 * alphaY * (pointInfoArr[threadIdx.y + 1][threadIdx.x].x - height);

	// Find F and G fluxes
	float3 fluxF = make_float3(height * momentumU, height*momentumU*momentumU + GRAVITY * height * height / 2.0f, height * momentumU * momentumV);
	float3 fluxG = make_float3(height * momentumV, height*momentumU*momentumV, height*momentumV*momentumV + GRAVITY * height * height / 2.0f);

	// Get the Jacobian Matrix

	
	fluxFArr[threadIdx.y][threadIdx.x] = fluxF;
	fluxGArr[threadIdx.y][threadIdx.x] = fluxG;
	
	__syncthreads(); // Fill block of threads with flux valeus

	// Find the 1st part of the low-order Lax Friedrich flux for the h component
	hFluxX = hFluxX + 0.5 * (fluxF.x + fluxFArr[threadIdx.y][threadIdx.x + 1].x);
	hFluxY = hFluxY + 0.5 * (fluxG.x + fluxGArr[threadIdx.y + 1][threadIdx.x].x);

	// Find the second order flux derivative wrt x and y
	float3 dfdx = (1.0f / (2.0f * dx)) * (fluxFArr[threadIdx.y][threadIdx.x + 1] - fluxFArr[threadIdx.y][threadIdx.x - 1]);
	float3 dfdy = (1.0f / (2.0f * dy)) * (fluxGArr[threadIdx.y + 1][threadIdx.x] - fluxGArr[threadIdx.y - 1][threadIdx.x]);

	// Find second order source term

	// Find F tilde and G tilde from paper, the time averaged fluxes
	float3 fTilde = fluxF ;// + (dt/2.0f) * multiplication term - check from paper
	float3 gTilde = fluxG ;// + (dt/2.0f) * multiplication term - check from paper

	// Replace the x and y component of pointInfoArr with the flux values, hFluxX and hFluxY
	pointInfoArr[threadIdx.y][threadIdx.x].x = hFluxX;
	pointInfoArr[threadIdx.y][threadIdx.x].y = hFluxY;

	__syncthreads(); // Update the point info arr with flux components of th  height

	// Finding Gamma for checks later
	float gammaVal = -(height - (dt / dx) * (hFluxX - pointInfoArr[threadIdx.y][threadIdx.x - 1].x) - (dt / dy) * (hFluxY - pointInfoArr[threadIdx.y - 1][threadIdx.x].y));



	/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

	// WENO Reconstruction in X Direction
	fluxFArr[threadIdx.y][threadIdx.x] = fTilde;
	fluxGArr[threadIdx.y][threadIdx.x] = make_float3(height + bathymetryVal, momentumU, momentumV); // Reuse shared memory to store point set quantities again

	__syncthreads(); // Fill shared memory with point information and F Flux values for X direction reconstruction

	// Set flux boundaries
	if((threadIdx.x + offsetX >= 0) && (threadIdx.x + offsetX < NUM_THREADS_X)){
		fluxFArr[threadIdx.y][threadIdx.x + offsetX] = fTilde * make_float3(-1.0f, 1.0f, 1.0f); // Reverse flux at boundaries
	}

	__syncthreads();

	float3 outFluxPosX, outFluxNegX;
	float outSourcePosX, outSourceNegX;

	// Applying Lax Friedrich flux splitting
	// Positive flux part
	float3 fPosX_iP_j = 0.5 * (fluxFArr[threadIdx.y][threadIdx.x - 1] + alphaX * fluxGArr[threadIdx.y][threadIdx.x - 1]); // iP denotes i previous
	float3 fPosX_i_j =   0.5 * (fluxFArr[threadIdx.y][threadIdx.x]     + alphaX * fluxGArr[threadIdx.y][threadIdx.x]    );
	float3 fPosX_iN_j = 0.5 * (fluxFArr[threadIdx.y][threadIdx.x + 1] + alphaX * fluxGArr[threadIdx.y][threadIdx.x + 1]); //iN dentoes i next

	// Reconstruct positive X
	WENOPosX(fPosX_iP_j, fPosX_i_j, fPosX_iN_j, terrainArr[threadIdx.y][threadIdx.x-1], terrainArr[threadIdx.y][threadIdx.x], terrainArr[threadIdx.y][threadIdx.x+1], outFluxPosX, outSourcePosX);

	// Negative flux part
	float3 fNegX_i_j = 0.5 * (fluxFArr[threadIdx.y][threadIdx.x] - alphaX * fluxGArr[threadIdx.y][threadIdx.x]); // iP denotes i previous
	float3 fNegX_iN_j =   0.5 * (fluxFArr[threadIdx.y][threadIdx.x + 1]     + alphaX * fluxGArr[threadIdx.y][threadIdx.x + 1]    );
	float3 fNegX_iNN_j = 0.5 * (fluxFArr[threadIdx.y][threadIdx.x + 2] + alphaX * fluxGArr[threadIdx.y][threadIdx.x + 2]); //iN dentoes i next

	// Reconstruct negative X
	WENONegX(fNegX_i_j, fNegX_iN_j, fNegX_iNN_j, terrainArr[threadIdx.y][threadIdx.x], terrainArr[threadIdx.y][threadIdx.x + 1], terrainArr[threadIdx.y][threadIdx.x+2], outFluxNegX, outSourceNegX);

	float3 outFluxX = outFluxNegX + outFluxPosX;
	float outSourceX = outSourcePosX + outSourceNegX;



	/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

	// WENO Reconstruction in Y Direction

	// Store G tilde for WENO reconstruction along Y direction
	pointInfoArr[threadIdx.y][threadIdx.x] = gTilde;
	__syncthreads();	

	// Set Flux boundaries
	if (((threadIdx.y + offsetY) >= 0) && ((threadIdx.y + offsetY) < NUM_THREADS_Y)) {
		pointInfoArr[threadIdx.y + offsetY][threadIdx.x] = gTilde * make_float3(-1.0f, 1.0f, 1.0f);
	}

	__syncthreads();


	float3 outFluxPosY, outFluxNegY;
	float outSourcePosY, outSourceNegY;

	// Applying Lax Friedrich flux splitting
	// Positive flux part
	float3 fPosY_i_jP = 0.5 * (fluxFArr[threadIdx.y - 1][threadIdx.x] + alphaY * fluxGArr[threadIdx.y - 1][threadIdx.x]); // iP denotes i previous
	float3 fPosY_i_j =   0.5 * (fluxFArr[threadIdx.y][threadIdx.x]     + alphaY * fluxGArr[threadIdx.y][threadIdx.x]);
	float3 fPosY_i_jN = 0.5 * (fluxFArr[threadIdx.y + 1][threadIdx.x] + alphaY * fluxGArr[threadIdx.y + 1][threadIdx.x]); //iN dentoes i next

	// Reconstruct positive X
	WENOPosY(fPosY_iP_j, fPosY_i_j, fPosY_iN_j, terrainArr[threadIdx.y - 1][threadIdx.x], terrainArr[threadIdx.y][threadIdx.x], terrainArr[threadIdx.y + 1][threadIdx.x], outFluxPosY, outSourcePosY);

	// Negative flux part
	float3 fNegY_i_j = 0.5 * (fluxFArr[threadIdx.y][threadIdx.x] - alphaY * fluxGArr[threadIdx.y][threadIdx.x]); // iP denotes i previous
	float3 fNegY_i_jN =   0.5 * (fluxFArr[threadIdx.y + 1][threadIdx.x] + alphaY * fluxGArr[threadIdx.y + 1][threadIdx.x]);
	float3 fNegY_i_jNN = 0.5 * (fluxFArr[threadIdx.y + 2][threadIdx.x] + alphaY * fluxGArr[threadIdx.y + 2][threadIdx.x]); //iN dentoes i next

	// Reconstruct negative X
	WENONegY(fNegY_i_j, fNegY_i_jN, fNegY_i_jNN, terrainArr[threadIdx.y][threadIdx.x], terrainArr[threadIdx.y + 1][threadIdx.x], terrainArr[threadIdx.y + 2][threadIdx.x], outFluxNegY, outSourceNegY);

	float3 outFluxY = outFluxNegY + outFluxPosY;
	float outSourceY = outSourcePosY + outSourceNegY;

}




/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////




// Constructor
SWE::SWE(int numPointsX, int numPointsY){

	this->numPointsX = numPointsX;
	this->numPointsY = numPointsY;
	// Allocate memory to host variables
	// Allocating height and momentum variables (+2 for boundary?)
	// U variable in Equation
	h_height = (float*)malloc(sizeof(float) * (numPointsX + 2) * (numPointsY + 2)); 
	h_momentumU = (float*)malloc(sizeof(float) * (numPointsX + 2) * (numPointsY + 2)); 
	h_momentumV = (float*)malloc(sizeof(float) * (numPointsX + 2) * (numPointsY + 2)); 

	// Allocating flux terms (why +1?)
	// F and G variables in formula
	h_Fh = (float*)malloc(sizeof(numPointsX + 1) * (numPointsY + 1)); 
	h_Fhu = (float*)malloc(sizeof(numPointsX + 1) * (numPointsY + 1));
	h_Fhv = (float*)malloc(sizeof(numPointsX + 1) * (numPointsY + 1));
	h_Gh = (float*)malloc(sizeof(numPointsX + 1) * (numPointsY + 1));
	h_Ghu = (float*)malloc(sizeof(numPointsX + 1) * (numPointsY + 1));
	h_Ghv = (float*)malloc(sizeof(numPointsX + 1) * (numPointsY + 1));

	// Allocating max height, max velocity and charachteristic velocity
	h_maxHeight = new float;
	h_maxVelocity = new float;
	h_characteristicVelocity = new float;

	// Allocating memory for offsets
	h_offsetX = (int*)malloc(sizeof(numPointsX) * (numPointsY)); 
	h_offsetY = (int*)malloc(sizeof(numPointsX) * (numPointsY)); 

	// Allocate memory to device variables
	checkCudaErrors(hipMalloc((void**)&d_height, (numPointsX+2)*(numPointsY+2)*sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&d_momentumU, (numPointsX+2)*(numPointsY+2)*sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&d_momentumV, (numPointsX+2)*(numPointsY+2)*sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&d_Fh, (numPointsX+1)*(numPointsY+1)*sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&d_Fhu, (numPointsX+1)*(numPointsY+1)*sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&d_Fhv, (numPointsX+1)*(numPointsY+1)*sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&d_Gh, (numPointsX+1)*(numPointsY+1)*sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&d_Ghu, (numPointsX+1)*(numPointsY+1)*sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&d_Ghv, (numPointsX+1)*(numPointsY+1)*sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&d_maxHeight, sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&d_maxVelocity, sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&d_characteristicVelocity, sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&d_offsetX, (numPointsX)*(numPointsY)*sizeof(int)));
	checkCudaErrors(hipMalloc((void**)&d_offsetY, (numPointsX)*(numPointsY)*sizeof(int)));


	// Initializing device variables
	checkCudaErrors(hipMemset(d_height, 0, (numPointsX+2)*(numPointsY+2)*sizeof(float)));
	checkCudaErrors(hipMemset(d_momentumU, 0, (numPointsX+2)*(numPointsY+2)*sizeof(float)));
	checkCudaErrors(hipMemset(d_momentumV, 0, (numPointsX+2)*(numPointsY+2)*sizeof(float)));
	checkCudaErrors(hipMemset(d_Fh, 0, (numPointsX+1)*(numPointsY+1)*sizeof(float)));
	checkCudaErrors(hipMemset(d_Gh, 0, (numPointsX+1)*(numPointsY+1)*sizeof(float)));
	checkCudaErrors(hipMemset(d_maxHeight, 0, sizeof(float)));
	checkCudaErrors(hipMemset(d_maxVelocity, 0, sizeof(float)));
	checkCudaErrors(hipMemset(d_characteristicVelocity, 0.001f, sizeof(float)));
	checkCudaErrors(hipMemset(d_offsetX, 0, (numPointsX)*(numPointsY)*sizeof(int)));
	checkCudaErrors(hipMemset(d_offsetY, 0, (numPointsX)*(numPointsY)*sizeof(int)));

}

// Set the starting conditions
void SWE::setInitialConditions(int conditionNum){
	switch(conditionNum){
		case 0:
			for(int i = 0; i < numPointsX; i++){
				for(int j = 0 ; j < numPointsY; j++){
					if(i > numPointsX/4 && i < 3*numPointsX/4 && j > numPointsY/4 && j < 3*numPointsY/4){
						h_height[i + j * (numPointsX + 2)] = 5.5f;
					}
					else{
						h_height[i + j * (numPointsX + 2)] = 1.0f;
				}
			}
		}
	}
}


