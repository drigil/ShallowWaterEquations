#include <iostream>
#include "SWE.cuh"

int main(){
	int numPointsX = 80;
	int numPointsY = 56;
	int conditionNum = 0;

	SWE swe(numPointsX, numPointsY);
	swe.setInitialConditions(conditionNum);

	for(int i = 0; i<numPointsX; i++){
		for(int j = 0; j<numPointsY; j++){
			printf("%f ", swe.h_height[i + j * (numPointsX)]);
		}
		printf(" \n");
	}
	
	swe.simulate();

	hipError_t kernelErr = hipGetLastError();
	if(kernelErr!=hipSuccess){
		printf("Error: %s\n", hipGetErrorString(kernelErr));
	}


	printf("After running simulation \n");
	for(int i = 0; i<numPointsX; i++){
		for(int j = 0; j<numPointsY; j++){
			printf("%f ", swe.h_height_out[i + j * (numPointsX)]);
		}
		printf(" \n");
	}
	
	
}